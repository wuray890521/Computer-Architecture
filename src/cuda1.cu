// cuda1.cu

#include "hip/hip_runtime.h"

__global__ void grayscaleConversion(unsigned char *inputImage, unsigned char *outputImage, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


    //allocate each pixel to threads 
    if (x < width && y < height) {
        int tid = y * width + x;
        unsigned char r = inputImage[3 * tid];
        unsigned char g = inputImage[3 * tid + 1];
        unsigned char b = inputImage[3 * tid + 2];
        outputImage[tid] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}




float GPU_kernel(unsigned char *outputImage_GPU,unsigned char *inputImage){

	

        int width = 32;
        int height = 32;
        size_t imageSize = width * height * 3 * sizeof(unsigned char);

 

	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop); 	

   
        //allocate GPU space
        unsigned char *d_inputImage, *d_outputImage_GPU;
        hipMalloc((void **)&d_inputImage, imageSize);
        hipMalloc((void **)&d_outputImage_GPU, width * height * sizeof(unsigned char));


        // CPU to GPU
        hipMemcpy(d_inputImage, inputImage, imageSize, hipMemcpyHostToDevice);

       // Start Timer
       hipEventRecord(start, 0);

        //define CUDA block and grid dimension
        dim3 blockDim(4, 4);
        dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

        //Use GPU to count
        grayscaleConversion<<<gridDim, blockDim>>>(d_inputImage, d_outputImage_GPU, width, height);

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop); 


        //GPU to CPU
        hipMemcpy(outputImage_GPU, d_outputImage_GPU, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);


	// Release Memory Space on Device
	hipFree(d_inputImage);
	hipFree(d_outputImage_GPU);
	//cudaFree(dInd);

	// Calculate Elapsed Time
  	float elapsedTime; 
  	hipEventElapsedTime(&elapsedTime, start, stop);  

	return elapsedTime;//retuen GPU time
}
