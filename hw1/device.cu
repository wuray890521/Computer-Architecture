#include "hip/hip_runtime.h"
#include "parameters.h"


__global__ void cuda_kernel(int *B,int *A,IndexSave *dInd)
{	
	// complete cuda kernel function	
};


float GPU_kernel(int *B,int *A,IndexSave* indsave){

	int *dA,*dB;
	IndexSave* dInd;

	// Creat Timing Event
  	hipEvent_t start, stop;
	hipEventCreate (&start);
	hipEventCreate (&stop); 	

	// Allocate Memory Space on Device

	// Allocate Memory Space on Device (for observation)
	hipMalloc((void**)&dInd,sizeof(IndexSave)*SIZE);

	// Copy Data to be Calculated

	// Copy Data (indsave array) to device
	hipMemcpy(dInd, indsave, sizeof(IndexSave)*SIZE, hipMemcpyHostToDevice);
	
	// Start Timer
	hipEventRecord(start, 0);

	// Lunch Kernel
	dim3 dimGrid(2);
	dim3 dimBlock(4);
	cuda_kernel<<<dimGrid,dimBlock>>>(dB,dA,dInd);

	// Stop Timer
	hipEventRecord(stop, 0);
  	hipEventSynchronize(stop); 

	// Copy Output back

	// Release Memory Space on Device
	hipFree(dA);
	hipFree(dB);
	hipFree(dInd);

	// Calculate Elapsed Time
  	float elapsedTime; 
  	hipEventElapsedTime(&elapsedTime, start, stop);  

	return elapsedTime;
}
